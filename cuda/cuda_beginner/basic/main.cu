#include <iostream>
#include<cstdlib>

#include <hipblas.h>
#include <hip/hip_runtime.h>

using namespace std;

#define N 1024

__global__ void add(int * a,int *b,int *c)
{
 c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
void random_ints(int* x, int size)
{
        int i;
        for (i=0;i<size;i++) {
                x[i]=rand()%10;
        }
}
int main() {
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    int size = N * sizeof(int);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);
    h_a = (int *)malloc(size);
    h_b = (int *) malloc(size);
    h_c = (int *) malloc(size);
    random_ints(h_a, N);
    random_ints(h_b, N);
    random_ints(h_c, N);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<N,1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
/*    for(int i=0;i<N;i++)
    {
       cout<< h_c[i]<<" ";
    }
    cout<<endl;
*/
    free(h_a);free(h_b);free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
//    cout <<"c= "<< h_c << endl;
    return 0;
}

