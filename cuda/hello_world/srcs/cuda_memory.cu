// Using different memory sapces in CUDA


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Using local memory
 */

// a __device__ or __global__ function runs on the GPU
__global__ void use_local_memory_GPU(float in)
{
	float f; // varible "f" is in local memory and private to each thread
	f = in;
}


/*
 * Using global memory
 */

// a __global__ function runs on the GPU & can be called from host
__global__ void use_global_memory_GPU(float* array)
{
	// "array" is a pointer into global memory on the device
	array[threadIdx.x] = 2.0f * (float)threadIdx.x;
}


/*
 * Using shared memory
 */

// for clarity, hardcoding 128 threads/elements and omitting out-of-bounds checks
__global__ void use_shared_memory_GPU(float *array)
{
	// local variables, private to each thread
	int i;
	int index = threadIdx.x;
	float average, sum = 0.0f;

	// __shared__ varibales are visible to all threads in the thread block
	// and have the same lifetime as the thread block
	__shared__ float sh_arr[128];

	// copy data from "array" in global memory to sh_arr in shared memory
	// here, each thread is responsible for copying a single element
	sh_arr[index] = array[index];

	// ensure all the writes wo shared memory have completed
	__syncthreads();

	// now, sh_arr is fully populated. Let's find the average of all previous elements
	for (int i = 0; i < index; i++)
		sum += sh_arr[i];
	average = sum / (index + 1.0f);

	// if array[index] is greater than the average of array[0..index-1], replace with average.
	// since array[] is in global memory, this change will be seen by the host (and potentially
	// other thread blocks, if any)
	if (array[index] > average)
		array[index] = average;

	// the following code has NO EFFECT: it modifies shared memory, but
	// the resulting modified data is never copied back to global memory
	// and vanishes when the thread block completes
	sh_arr[index] = 3.14;
}

int main(int argc, char** argv)
{
	/*
	 * First, call a kernel that shows using local memory
	 */
	use_local_memory_GPU<<<1, 128>>>(2.0f);

	/*
	 * Next, call a kernel that shows using global memory
	 */
	float h_arr[128]; // convention: h_ variables live on host
	float *d_arr;	  // convention: d_ variables live on device (GPU global mem)

	// allocate global memory on the device, place result in "d_arr"
	hipMalloc((void**)&d_arr, sizeof(float) * 128);
	// now copy data from host memory to device memory
	hipMemcpy((void *)d_arr, (void*)h_arr, sizeof(float) * 128, hipMemcpyHostToDevice);
	// launch the kernel
	use_global_memory_GPU<<<1, 128>>>(d_arr);
	// copy the modified array back to the host, overwriting contents of h_arr
	hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);
	// ... do other stuff ...
	for (int i = 0; i < 128; ++i)
	{
		printf("%f", h_arr[i]);
		printf((i % 4 != 3) ? "\t" : "\n");
	}

	/*
	 * At last, call a kernel that show using shared memory
	 */

	// as before, pass in a pointer to data in global memory
	use_shared_memory_GPU<<<1, 128>>>(d_arr);
	// copy the modified array back to the host
	hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);

	for (int i = 0; i < 128; ++i)
	{
		printf("%f", h_arr[i]);
		printf((i % 4 != 3) ? "\t" : "\n");
	}

	return 0;
}