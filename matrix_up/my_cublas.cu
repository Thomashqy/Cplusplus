#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include<time.h>
#include<stdlib.h>


void print_matrix(float* mat, int row, int col) {
    for (int i = 0; i < row * col; i++) {
        printf("%f\t", mat[i]);
        if ((i+1) % col == 0) {
            printf("\n");
        }
        
    }
     printf("----------------------------\n");
}


int main(int argc, char *argv[]) {
    float *mat1, *mat2, *result;
    float *g_mat1, *g_mat2, *g_mat_result;
    int r_size, m_size;    // 矩阵行数，矩阵size

    hipError_t cudaStat;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    if (argc > 1) {
        r_size = atoi(argv[1]);
    } else {
        r_size = 8192;
    }
    m_size = r_size * r_size;
    
    // 用一位数组表示二维矩阵
    mat1 = (float*) malloc(m_size * sizeof(float));
    mat2 = (float*) malloc(m_size * sizeof(float));
    result = (float*) malloc(m_size * sizeof(float));

    // initialize
    for (int i = 0; i < m_size; i++) {
        mat1[i] = rand()/10000000;
        mat2[i] = rand()/10000000;
        result[i] = 0; 
    }
    clock_t start,finish;
    
    start = clock();
    cudaStat = hipMalloc((void **)&g_mat1, sizeof(*mat1) * m_size);
    cudaStat = hipMalloc((void **)&g_mat2, sizeof(*mat2) * m_size);
    cudaStat = hipMalloc((void **)&g_mat_result, sizeof(*result) * m_size);
    printf("cudaStat %d\n", cudaStat);

    // initialize CUBLAS context
    stat = hipblasCreate(&handle);

    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat1), mat1, r_size, g_mat1, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat2), mat2, r_size, g_mat2, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*result), result, r_size, g_mat_result, r_size);

    float al = 1.0f;
    float bet = 0.0f;
    
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        r_size, r_size, r_size, &al, g_mat1, 
        r_size, g_mat2, r_size, &bet, g_mat_result, r_size);
    stat = hipblasGetMatrix(r_size, r_size, sizeof(*result), g_mat_result, r_size, result, r_size);
    printf("cublas %d\n",stat);
    hipMemcpy(result, g_mat_result, sizeof(float) * m_size, hipMemcpyDeviceToHost);
    
    finish = clock();
    printf("total times: %.3f\n",(double)(finish-start)/CLOCKS_PER_SEC);
    if (r_size < 10) {
        printf("-----mat1----\n");
        print_matrix(mat1, r_size, r_size);
        printf("-----mat2----\n");
        print_matrix(mat2, r_size, r_size);
        printf("----mat1 * mat2---\n");
        print_matrix(result, r_size, r_size);
    }
    for(int i=0;i<10;i++)
       printf("%.2f ",result[i]);
    printf("\n");
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
    free(mat1);
    free(mat2);
    free(result);
}
